#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudawarping.hpp>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <cstdio>

#define CUDA_CHECK(err)                                                                    \
    do                                                                                     \
    {                                                                                      \
        hipError_t e = (err);                                                             \
        if (e != hipSuccess)                                                              \
        {                                                                                  \
            fprintf(stderr, "CUDA %s:%d %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                                            \
        }                                                                                  \
    } while (0)

/* ================= CUDA kernels ================= */

/* 第一次扫描：4-邻域等价标记 */
__global__ void first_scan(const uint8_t *mask,
                           int *labels,
                           int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    const int idx = y * width + x;
    if (mask[idx] == 0)
    {
        labels[idx] = 0;
        return;
    }

    /* 上、左根标签 */
    int up = (y > 0) ? labels[idx - width] : 0;
    int left = (x > 0) ? labels[idx - 1] : 0;

    if (up == 0 && left == 0)
        labels[idx] = idx + 1; // 新标签
    else if (up == 0)
        labels[idx] = left;
    else if (left == 0)
        labels[idx] = up;
    else
        labels[idx] = min(up, left); // 合并
}

/* 第二次扫描：路径压缩 */
__global__ void second_scan(int *labels, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    int lab = labels[idx];
    if (lab == 0)
        return;

    while (labels[lab - 1] != lab)
        lab = labels[lab - 1];
    labels[idx] = lab;
}

/* 只保留最大域 */
__global__ void keep_max(const int *labels,
                         uint8_t *dst,
                         int max_label,
                         int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    dst[idx] = (labels[idx] == max_label) ? 0 : 1;
}
/* 种子填充：迭代扫描，8 邻域 */
__global__ void flood_fill_step(uint8_t *mask, int w, int h, bool *changed)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h)
        return;

    const int idx = y * w + x;
    if (mask[idx] != 0)
        return; // 已经是前景跳过

    /* 8 邻域只要有一个背景点 → 仍是背景 */
    for (int dy = -1; dy <= 1; ++dy)
        for (int dx = -1; dx <= 1; ++dx)
        {
            int nx = x + dx, ny = y + dy;
            if (nx < 0 || ny < 0 || nx >= w || ny >= h)
                continue;
            if (mask[ny * w + nx] == 0)
                return;
        }
    /* 所有邻域都是前景 → 该点置前景 */
    mask[idx] = 1;
    *changed = true;
}
/* ================= 主接口 ================= */
cv::Mat remove_small_white_regions_cuda1(const cv::Mat &src)
{
    CV_Assert(src.type() == CV_8UC1);
    const int w = src.cols;
    const int h = src.rows;
    const int n = w * h;

    /* GPU 上传 */
    cv::cuda::GpuMat d_src, d_tmp, d_labels;
    d_src.upload(src);
    d_src.copyTo(d_tmp); // 工作副本，用于孔洞填充
    d_labels.create(h, w, CV_32SC1);

    dim3 block(32, 16);
    dim3 grid((w + block.x - 1) / block.x,
              (h + block.y - 1) / block.y);

    /* ---------- 1. 填充孔洞 ---------- */
    bool *d_changed;
    CUDA_CHECK(hipMalloc(&d_changed, sizeof(bool)));
    do
    {
        bool h_changed = false;
        CUDA_CHECK(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
        flood_fill_step<<<grid, block>>>(d_tmp.ptr<uint8_t>(), w, h, d_changed);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
        if (!h_changed)
            break;
    } while (true);
    CUDA_CHECK(hipFree(d_changed));

    /* ---------- 2. 连通域标记 ---------- */
    first_scan<<<grid, block>>>(d_tmp.ptr<uint8_t>(),
                                reinterpret_cast<int *>(d_labels.ptr<uint8_t>()),
                                w, h);
    CUDA_CHECK(hipDeviceSynchronize());
    second_scan<<<grid, block>>>(reinterpret_cast<int *>(d_labels.ptr<uint8_t>()),
                                 w, h);
    CUDA_CHECK(hipDeviceSynchronize());

    /* ---------- 3. 统计最大域 ---------- */
    thrust::device_ptr<int> d_lab(reinterpret_cast<int *>(d_labels.ptr<uint8_t>()));
    thrust::device_vector<int> labs(d_lab, d_lab + n);
    thrust::sort(labs.begin(), labs.end());

    thrust::device_vector<int> keys(n);
    thrust::device_vector<int> areas(n);
    auto new_end = thrust::reduce_by_key(labs.begin(), labs.end(),
                                         thrust::constant_iterator<int>(1),
                                         keys.begin(),
                                         areas.begin());

    int num_labels = new_end.second - areas.begin();
    auto max_it = thrust::max_element(areas.begin(), areas.begin() + num_labels);
    int max_label = keys[max_it - areas.begin()];

    /* ---------- 4. 只保留最大域 ---------- */
    keep_max<<<grid, block>>>(reinterpret_cast<int *>(d_labels.ptr<uint8_t>()),
                              d_tmp.ptr<uint8_t>(),
                              max_label,
                              w, h);
    CUDA_CHECK(hipDeviceSynchronize());

    cv::Mat dst;
    d_tmp.download(dst);
    return dst;
}